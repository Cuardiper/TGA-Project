#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#define STB_IMAGE_IMPLEMENTATION
#include <math.h>
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <time.h>
#include <dirent.h>

#define SIZE 32


void read_frames(uint8_t* frame, int size, int sizeFrame) {
	for (int i = 0; i < size; ++i) {
		char filename[300];
		sprintf(filename, "pics/thumb%d.jpg",i+1);
		int width, height, bpp;
		uint8_t* rgb_image = stbi_load(filename, &width, &height, &bpp, 3);
        uint8_t he = height;
        uint8_t wi = width;
        frame[i*sizeFrame] = he;
        frame[i*sizeFrame+1] = wi;
        frame[i*sizeFrame+2] = bpp;
        for(int j = 0; j < height*width*3; ++j)
        frame[i*sizeFrame+3+j] = rgb_image[j];
    }
}

int max(int n1) {
	return n1>255 ? 255 : n1;
}


////////////////////  |
///CODIGO CUDA//////  |
///////////////////   v

__global__ void KernelByN (int Nfil, int Ncol, uint8_t *Input, uint8_t *Output, float *kernel, int Nframes, int SzFrame, int szKernel) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int R,G,B;
    if(row < Nfil && col < Ncol) {
        for (int i = 0; i < Nframes; ++i) {
            int ind = (row * Ncol + col)*3 + i*SzFrame + 3;
            R=G=B=0;
            for (int m = 0; m < szKernel; ++m)			// kernel rows
            {

                for (int n = 0; n < szKernel; ++n)		// kernel columns	
                {

                    // index of input signal used for checking boundary
                    int ii = row + (2 - m);
                    int jj = col + (2 - n);
                    
                    //ignore input samples which are out of bound
                    if(ii >= 0 && ii < Nfil && jj >= 0 && jj < Ncol){
                        R += Input[i*SzFrame + 3 + (ii*Ncol + jj)*3] * kernel[m*szKernel+n];
                        G += Input[i*SzFrame + 3 + ((ii*Ncol + jj)*3)+1] * kernel[m*szKernel+n];
                        B += Input[i*SzFrame + 3 + ((ii*Ncol + jj)*3)+2] * kernel[m*szKernel+n];
                    }
                }
            }
            Output[ind] = R; 
            Output[ind+1] = G;
            Output[ind+2] = B;
        }
    }
}

void CheckCudaError(char sms[], int line);



int main(int argc, char** argv)
{
    
  if (argc < 2) {
		printf("Necesito la ruta del video en mp4!\n");
		return -1;
	}
	int Nfil, Ncol;
	unsigned long numBytes;
	unsigned int nThreads;

	float TiempoTotal, TiempoKernel;
	hipEvent_t E0, E1, E2, E3;
//     float KH[9] = {0,-1,0,-1,5,-1,0,-1,0};
    float KH[3][3] = {{(float)1/16,(float)1/8,(float)1/16}, {(float)1/8,(float)1/4,(float)1/8}, {(float)1/16,(float)1/8,(float)1/16}};
//     static float *KH = 
	uint8_t *Host_I;
	uint8_t *Host_O;
	uint8_t *Dev_I;
	uint8_t *Dev_O;
	float *Kernel;

	//Sacar los fotogramas del video usando FFMPEG
    char *filename = argv[1];
//     system("mkdir pics");
    system("mkdir pics2");
    char *auxCommand = "pics/thumb%d.jpg -hide_banner";
    char comando[300];
    sprintf(comando, "ffmpeg -i %s.mp4 %s",filename,auxCommand);
//     system(comando);
    sprintf(comando,"ffmpeg -i %s.mp4 -vn -acodec copy audio.aac",filename);
    system(comando);

	//Contar el numero de fotogramas obtenidos
	DIR *d;
	struct dirent *dir;
	d = opendir("pics/");
	unsigned long frames = 0;
	if (d) {
		while ((dir = readdir(d)) != NULL) {
			frames++;
		}
		closedir(d);
	}

    int bpp;
    stbi_load("pics/thumb1.jpg", &Nfil, &Ncol, &bpp, 3);
	printf("Leyendo %d fotogramas de %d x %d resolucion...\n",frames-2, Nfil, Ncol);
    Nfil = Nfil * 3;

    numBytes = (frames-2) * (3 + Nfil * Ncol) * sizeof(uint8_t); //Guardamos 3 uint8_t (height, width i bpp) + un uint8_t por cada color (3*width*height)
    //Podemos cargarnos la struct y considerar que los 3 primeros valores son height, width y bpp, y los (3*width*height) siguientes el data, todo eso por cada frame.
    //Cada frame ocupa 3*Nfil*Ncol uint8_t.

    // Obtener Memoria en el host
    printf("Numero de bytes: %lu\n", numBytes);
    Host_I = (uint8_t*) malloc(numBytes);
    if(Host_I == NULL)
    {
        printf("Memory allocation failed\n");
        return;
    }
    Host_O = (uint8_t*) malloc(numBytes);
    if(Host_O == NULL)
    {
        printf("Memory allocation failed\n");
        return;
    }
    read_frames(Host_I, frames-2, 3 + Nfil * Ncol);

	hipEventCreate(&E0);	hipEventCreate(&E1);
    hipEventCreate(&E2);	hipEventCreate(&E3);
    printf("Aplicando filtro...\n");
    //
    // KERNEL ELEMENTO a ELEMENTO
    //

    // numero de Threads en cada dimension 
    nThreads = SIZE;

	// numero de Blocks en cada dimension
	int nBlocksFil = (Nfil/3+nThreads-1)/nThreads; //tener en cuenta 3componentes RGB??
	int nBlocksCol = (Ncol+nThreads-1)/nThreads;
    

	dim3 dimGridE(nBlocksCol, nBlocksFil, 1);
	dim3 dimBlockE(nThreads, nThreads, 1);
    
    hipEventRecord(E0, 0);
    hipEventSynchronize(E0);
    // Obtener Memoria en el device
    hipMalloc((uint8_t**)&Dev_I, numBytes);
    hipMalloc((uint8_t**)&Dev_O, numBytes);
    hipMalloc((float**)&Kernel, 3*3*sizeof(float));
    // Copiar datos desde el host en el device 
    hipMemcpy(Dev_I, Host_I, numBytes, hipMemcpyHostToDevice);
    CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);
    hipMemcpy(Dev_O, Host_O, numBytes, hipMemcpyHostToDevice);
    CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);
    hipMemcpy(Kernel, KH, 9*sizeof(float), hipMemcpyHostToDevice);
    CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);
        
    hipEventRecord(E1, 0);
    hipEventSynchronize(E1);
	// Ejecutar el kernel elemento a elemento
	KernelByN<<<dimGridE, dimBlockE>>>(Nfil/3, Ncol, Dev_I, Dev_O, Kernel, frames-2, 3 + Nfil * Ncol, 3);
	CheckCudaError((char *) "Invocar Kernel", __LINE__);

	hipEventRecord(E2, 0);
	hipEventSynchronize(E2);

	// Obtener el resultado desde el host
	hipMemcpy(Host_O, Dev_O, numBytes, hipMemcpyDeviceToHost);
	CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);

	// Liberar Memoria del device 
	hipFree(Dev_I);
	hipFree(Dev_O);
	hipFree(Kernel);
    hipEventRecord(E3, 0);
    hipEventSynchronize(E3);

    hipEventElapsedTime(&TiempoTotal,  E0, E3);
    hipEventElapsedTime(&TiempoKernel, E1, E2);
    printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
    printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);
    printf("Bandwidth: %4.6f GB/s\n", (float)(((float)(numBytes/TiempoKernel))/1000000));
    printf("Rendimiento Global: %4.2f GFLOPS\n", (3.0 * (float) Nfil/3 * (float) Ncol * (float) (frames-2)) / (1000000.0 * TiempoTotal));
    printf("Rendimiento Kernel: %4.2f GFLOPS\n", (3.0 * (float) Nfil/3 * (float) Ncol * (float) (frames-2)) / (1000000.0 * TiempoKernel));
	hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);
    printf("Writing...\n");
    char picname[300];
    for (int i = 0; i < frames-2; ++i) {
        printf("\rIn progress %d", i*100/(frames-2)); ///'size' no definido (solución: lo pongo en mayusculas, no se si es la variable a la que te querias referir)
        sprintf(picname, "thumb%d.jpg",i+1);
        char ruta [300];
        sprintf(ruta, "pics2/%s",picname);
        stbi_write_jpg(ruta, Nfil/3, Ncol, 3, &Host_O[i*(3 + Nfil * Ncol)+3], Nfil);   //He cambiado out[] por Host_O[]
    }
    auxCommand = "ffmpeg -framerate 25 -i pics2/thumb%d.jpg";
	sprintf(comando, "%s -pattern_type glob -c:v libx264 -pix_fmt yuv420p %s_out_provisional.mp4",auxCommand, filename);
	system(comando);
	sprintf(comando,"ffmpeg -i %s_out_provisional.mp4 -i audio.aac -c:v copy -c:a aac -strict experimental %s_out.mp4",filename,filename);
	system(comando);
	sprintf(comando,"rm %s_out_provisional.mp4",filename);
	system(comando);
	system("rm audio.aac");
	system("rm -rf pics2");
    return 0;
    
}

void CheckCudaError(char sms[], int line) {
  hipError_t error;

  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }
  //else printf("(OK) %s \n", sms);
}
