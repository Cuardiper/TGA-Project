#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#define STB_IMAGE_IMPLEMENTATION
#include <math.h>
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <time.h>
#include <dirent.h>

#define SIZE 32


void read_frames(uint8_t* frame, int size, int sizeFrame) {
	for (int i = 0; i < size; ++i) {
		char filename[300];
		sprintf(filename, "pics/thumb%d.jpg",i+1);
		int width, height, bpp;
		uint8_t* rgb_image = stbi_load(filename, &width, &height, &bpp, 3);
        uint8_t he = height;
        uint8_t wi = width;
        frame[i*sizeFrame] = he;
        frame[i*sizeFrame+1] = wi;
        frame[i*sizeFrame+2] = bpp;
        for(int j = 0; j < height*width*3; ++j)
            frame[i*sizeFrame+3+j] = rgb_image[j];
    }
}


////////////////////  |
///CODIGO CUDA//////  |
///////////////////   v

__global__ void KernelByN (int Nfil, int Ncol, uint8_t *A, int Nframes, int SzFrame) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    
    if(row < Nfil && col < Ncol){
        for (int i = 0; i < Nframes; ++i) {
            int ind = (row * Ncol + col)*3 + i*SzFrame + 3;
            A[ind] = A[ind+1] = A[ind+2] = (A[ind] + A[ind+1] + A[ind+2])/3 > 127 ? (uint8_t) 255 : (uint8_t) 0;
        }
    }
}

void CheckCudaError(char sms[], int line);




int main(int argc, char** argv)
{
    
  if (argc < 2) {
		printf("Necesito la ruta del video en mp4!\n");
		return -1;
	}
	int Nfil, Ncol;
	unsigned long numBytes;
	unsigned int nThreads;

	float TiempoTotal, TiempoKernel;
	hipEvent_t E0, E1, E2, E3;

	uint8_t *Host_I;
	uint8_t *Host_O;
	uint8_t *Dev_I;

	//Sacar los fotogramas del video usando FFMPEG
    char *filename = argv[1];
//     system("mkdir pics");
    system("mkdir pics2");
    char *auxCommand = "pics/thumb%d.jpg -hide_banner";
    char comando[300];
    sprintf(comando, "ffmpeg -i %s.mp4 %s",filename,auxCommand);
//     system(comando);
    sprintf(comando,"ffmpeg -i %s.mp4 -vn -acodec copy audio.aac",filename);
    system(comando);

	//Contar el numero de fotogramas obtenidos
	DIR *d;
	struct dirent *dir;
	d = opendir("pics/");
	unsigned long frames = 0;
	if (d) {
		while ((dir = readdir(d)) != NULL) {
			frames++;
		}
		closedir(d);
	}

    int bpp;
    stbi_load("pics/thumb1.jpg", &Nfil, &Ncol, &bpp, 3);
	printf("Leyendo %d fotogramas de %d x %d resolucion...\n",frames-2, Nfil, Ncol);
    Nfil = Nfil * 3;

    numBytes = (frames-2) * (3 + Nfil * Ncol) * sizeof(uint8_t); //Guardamos 3 uint8_t (height, width i bpp) + un uint8_t por cada color (3*width*height)
    //Podemos cargarnos la struct y considerar que los 3 primeros valores son height, width y bpp, y los (3*width*height) siguientes el data, todo eso por cada frame.
    //Cada frame ocupa 3*Nfil*Ncol uint8_t.

    // Obtener Memoria en el host
    printf("Numero de bytes: %lu\n", numBytes);
    Host_I = (uint8_t*) malloc(numBytes);
    if(Host_I == NULL)
    {
        printf("Memory allocation failed\n");
        return;
    }
    Host_O = (uint8_t*) malloc(numBytes);
    if(Host_O == NULL)
    {
        printf("Memory allocation failed\n");
        return;
    }
    read_frames(Host_I, frames-2, 3 + Nfil * Ncol);

	hipEventCreate(&E0);	hipEventCreate(&E1);
    hipEventCreate(&E2);	hipEventCreate(&E3);
    printf("Aplicando filtro...\n");
    //
    // KERNEL ELEMENTO a ELEMENTO
    //

    // numero de Threads en cada dimension 
    nThreads = SIZE;

	// numero de Blocks en cada dimension
	int nBlocksFil = (Nfil+nThreads-1)/nThreads; //tener en cuenta 3componentes RGB??
	int nBlocksCol = (Ncol+nThreads-1)/nThreads;
    

	dim3 dimGridE(nBlocksCol, nBlocksFil, 1);
	dim3 dimBlockE(nThreads, nThreads, 1);
    
    hipEventRecord(E0, 0);
    hipEventSynchronize(E0);
    // Obtener Memoria en el device
    hipMalloc((uint8_t**)&Dev_I, numBytes);
    // Copiar datos desde el host en el device 
    hipMemcpy(Dev_I, Host_I, numBytes, hipMemcpyHostToDevice);
    CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);
        
    hipEventRecord(E1, 0);
    hipEventSynchronize(E1);
	// Ejecutar el kernel elemento a elemento
	KernelByN<<<dimGridE, dimBlockE>>>(Nfil/3, Ncol, Dev_I, frames-2, 3 + Nfil * Ncol);
	CheckCudaError((char *) "Invocar Kernel", __LINE__);

	hipEventRecord(E2, 0);
	hipEventSynchronize(E2);

	// Obtener el resultado desde el host
	hipMemcpy(Host_I, Dev_I, numBytes, hipMemcpyDeviceToHost);
	CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);

	// Liberar Memoria del device 
	hipFree(Dev_I);
    hipEventRecord(E3, 0);
    hipEventSynchronize(E3);

    hipEventElapsedTime(&TiempoTotal,  E0, E3);
    hipEventElapsedTime(&TiempoKernel, E1, E2);
    printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
    printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);
    printf("Bandwidth: %4.6f GB/s\n", (float)(((float)(numBytes/TiempoKernel))/1000000));
    printf("Rendimiento Global: %4.2f GFLOPS\n", (3.0 * (float) Nfil/3 * (float) Ncol * (float) (frames-2)) / (1000000.0 * TiempoTotal));
    printf("Rendimiento Kernel: %4.2f GFLOPS\n", (3.0 * (float) Nfil/3 * (float) Ncol * (float) (frames-2)) / (1000000.0 * TiempoKernel));
	hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);
    printf("Writing...\n");
    char picname[300];
    for (int i = 0; i < frames-2; ++i) {
        printf("\rIn progress %d", i*100/(frames-2)); ///'size' no definido (solución: lo pongo en mayusculas, no se si es la variable a la que te querias referir)
        sprintf(picname, "thumb%d.jpg",i+1);
        char ruta [300];
        sprintf(ruta, "pics2/%s",picname);
        stbi_write_jpg(ruta, Nfil/3, Ncol, 3, &Host_I[i*(3 + Nfil * Ncol)+3], Nfil);   //He cambiado out[] por Host_O[]
    }
    auxCommand = "ffmpeg -framerate 25 -i pics2/thumb%d.jpg";
	sprintf(comando, "%s -pattern_type glob -c:v libx264 -pix_fmt yuv420p %s_out_provisional.mp4",auxCommand, filename);
	system(comando);
	sprintf(comando,"ffmpeg -i %s_out_provisional.mp4 -i audio.aac -c:v copy -c:a aac -strict experimental %s_out.mp4",filename,filename);
	system(comando);
	sprintf(comando,"rm %s_out_provisional.mp4",filename);
	system(comando);
	system("rm audio.aac");
	system("rm -rf pics2");
    return 0;
    
}

void CheckCudaError(char sms[], int line) {
  hipError_t error;

  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }
  //else printf("(OK) %s \n", sms);
}
